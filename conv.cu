#include "hip/hip_runtime.h"

__global__ void conv(pixel ** p,pixel *new,int width,int height,int size)
{

  x = threadIdx.x+blockIdx.x*BLOCK_SIZE;
  y = threadIdx.y+blockIdx.y*BLOCK_SIZE;
  int stencil_j, stencil_k ;
  int t_r = 0 ;
  int t_g = 0 ;
  int t_b = 0 ;

  for ( stencil_j = -size ; stencil_j <= size ; stencil_j++ )
  {
      for ( stencil_k = -size ; stencil_k <= size ; stencil_k++ )
      {
          t_r += p[i][CONV(j+stencil_j,k+stencil_k,width)].r ;
          t_g += p[i][CONV(j+stencil_j,k+stencil_k,width)].g ;
          t_b += p[i][CONV(j+stencil_j,k+stencil_k,width)].b ;
      }
  }

  new[CONV(j,k,width)].r = t_r / ( (2*size+1)*(2*size+1) ) ;
  new[CONV(j,k,width)].g = t_g / ( (2*size+1)*(2*size+1) ) ;
  new[CONV(j,k,width)].b = t_b / ( (2*size+1)*(2*size+1) ) ;


}

void
apply_blur_filter( animated_gif * image, int size, int threshold )
{
    int i, j, k ;
    int width, height ;
    int end = 0 ;
    int n_iter = 0 ;

    pixel ** p ;
    pixel * new ;

    /* Get the pixels of all images */
    p = image->p ;


    /* Process all images */
    for ( i = 0 ; i < image->n_images ; i++ )
    {
        n_iter = 0 ;
        width = image->width[i] ;
        height = image->height[i] ;

        /* Allocate array of new pixels */
        new = (pixel *)malloc(width * height * sizeof( pixel ) ) ;
        size_p=width * height * sizeof( pixel )*image->n_images ;
        size_new=width * height * sizeof( pixel ) ;
        /* Perform at least one blur iteration */
        pixel ** p_gpu; hipMalloc(&p_gpu,size_p);
        pixel * new_gpu; hipMalloc(&new_gpu,size_new);

        hipMemcpy(p_gpu,p,size_p,hipMemcpyHostToDevice);
        hipMemcpy(new_gpu,new,size_new,hipMemcpyHostToDevice);
        do
        {
            end = 1 ;
            n_iter++ ;

            /* Apply blur on top part of image (10%) */
            conv<<<1024, 256>>>(p_gpu,new_gpu, width,height,size);


            hipMemcpy(p,p_gpu,size_p,hipMemcpyDeviceToHost);
            hipMemcpy(new,new_gpu,size_new,hipMemcpyDeviceToHost);
            //#pragma omp parallel for omp_set_num_threads(7) schedule(dynamic)
            for(j=1; j<height-1; j++)
            {
                for(k=1; k<width-1; k++)
                {

                    float diff_r ;
                    float diff_g ;
                    float diff_b ;

                    diff_r = (new[CONV(j  ,k  ,width)].r - p[i][CONV(j  ,k  ,width)].r) ;
                    diff_g = (new[CONV(j  ,k  ,width)].g - p[i][CONV(j  ,k  ,width)].g) ;
                    diff_b = (new[CONV(j  ,k  ,width)].b - p[i][CONV(j  ,k  ,width)].b) ;

                    if ( diff_r > threshold || -diff_r > threshold
                            ||
                             diff_g > threshold || -diff_g > threshold
                             ||
                              diff_b > threshold || -diff_b > threshold
                       ) {
                        end = 0 ;
                    }

                    p[i][CONV(j  ,k  ,width)].r = new[CONV(j  ,k  ,width)].r ;
                    p[i][CONV(j  ,k  ,width)].g = new[CONV(j  ,k  ,width)].g ;
                    p[i][CONV(j  ,k  ,width)].b = new[CONV(j  ,k  ,width)].b ;
                }
            }

        }
        while ( threshold > 0 && !end ) ;

        // printf( "Nb iter for image %d\n", n_iter ) ;
        hipFree(p_gpu); hipFree(new_gpu);

        free (new) ;
    }

}
